#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include "psimulation.cuh"
#include <chrono>
#include <fstream>
using namespace std;

int main()
{
    int num_particles = 1024;
    float3 box_origin = make_float3(0.0f, 0.0f, 0.0f);
    float3 box_size = make_float3(4.8f, 4.8f, 4.8f);
    float cutoff_radius = 1.0f;
    float skin = 0.2*cutoff_radius;
    float cell_size = cutoff_radius + skin;
    int num_steps = 1000000;
    float dt = 0.01f;
    float force_strength = 1.0f;
    float force_r0 = 0.5f*cutoff_radius;
    float force_alpha = 5.0f/cutoff_radius;
    default_random_engine eng(42);
    ParticleSimulation sim(num_particles, force_strength, force_r0, force_alpha, box_origin, box_size, cell_size, cutoff_radius);
    sim.random_init(eng, box_origin, box_origin + box_size);
    sim.set_threads_per_block(64);
    float total_time = 0;
    auto time_start = std::chrono::high_resolution_clock::now();
    for(int step = 0; step < num_steps; step++) {
        sim.updateCellList();
        sim.loop();
        sim.step(dt);
        if(step % 10000 == 0) {
            cout << "Step " << step << "\n";
        }
    };
    auto time_end = std::chrono::high_resolution_clock::now();
    total_time = std::chrono::duration<float, std::milli>(time_end - time_start).count();
    cout << "Average time per step: " << total_time/num_steps << " ms\n";

    
    Particle* h_particles = new Particle[num_particles];
    hipMemcpy(h_particles, sim.particles, sizeof(Particle) * num_particles, hipMemcpyDeviceToHost);
    ofstream outfile("particles.txt");
    for(int i = 0; i < num_particles; i++) {
        outfile << h_particles[i].position.x << " " << h_particles[i].position.y << " " << h_particles[i].position.z;
        outfile << " " << h_particles[i].polarity.x << " " << h_particles[i].polarity.y << " " << h_particles[i].polarity.z << "\n";
    };
    outfile.close();
    delete[] h_particles;
    sim.free();
    return 0;
};